/* -*- mode: C -*- */
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

typedef unsigned int uint32;

__device__ uint32 
leftRotate( uint32 x, uint32 c ) {
    return ( x << c ) | ( x >> (32 - c)); 
}


/* Assume trailer is 1 word only. */
__device__ uint32
getMessageWord( int g, uint32 *nonzeroWords, int nonzeroLen, uint32 trailer ) {
    if ( g < nonzeroLen ) {
        return nonzeroWords[g];
    } else if ( g == 14 ) {
        /* Length is a 64-bit quantity stored LSB first */
        return trailer;
    } else {
        return 0;
    }
}


#define SHIFT do {                                                         \
        uint32 dTemp = d;                                               \
        d = c;                                                          \
        c = b;                                                          \
        b = b + leftRotate( a + f + K[i] + getMessageWord( g, nonzeroWords, numWords, originalLen ), shiftTable[i] ); \
        a = dTemp;                                                      \
    } while ( false )
 
inline __device__ int
countLeadingZeros( uint32 a ) {
    return __clz( __byte_perm( a, 0, 0x0123 ) );
}

/**
 * Calculate MD5 of "fixedPortion", a numeric suffix, and zero-padding.
 * Check for at least numZeros 
 */
__global__ void
md5Kernel( int *shiftTable, uint32 *K, char *fixedPortion, int fixedLen, int goalMask, int startN, int *goal ) {
    const uint32 a0 = 0x67452301;
    const uint32 b0 = 0xefcdab89;
    const uint32 c0 = 0x98badcfe;
    const uint32 d0 = 0x10325476;
    
    uint32 i;

    /* The grid is a one-dimensional array of one-dimensional blocks. */
    uint32 n = startN + blockDim.x * blockIdx.x + threadIdx.x;

    /* Calculate log base 10 of n, and build a string that long */
    char numeric[12];
    int pos = 11;
    int val = n;
    for ( ; pos >= 0; --pos ) {
        numeric[pos] = '0' + ( val % 10 );
        val /= 10;
        if ( val == 0 ) break;
    }
    /* pos was the last character written. */
    int numberLen = 12 - pos;
    int numBytes = fixedLen + numberLen + 1;
    int numWords = ( numBytes + 3 ) / 4; /* Round up to the nearest 32-bit word. */
    uint32 nonzeroWords[ 15 ];
    nonzeroWords[numWords-1] = 0; /* Ensure zero bits at the end */

    /* Arrange the nonzero portion of the 512-bit chunk */
    char * nonzeroBytes = (char *)nonzeroWords;
    memcpy( nonzeroBytes, fixedPortion, fixedLen );
    memcpy( nonzeroBytes + fixedLen, numeric + pos, numberLen );
    nonzeroBytes[fixedLen + numberLen] = 0x80; /* 1 bit in MSB required */

    int originalLen = ( numBytes - 1 ) * 8;

    uint32 a = a0, b = b0, c = c0, d = d0;
    for ( i = 0; i <= 15; ++i ) {
        uint32 f = ( b & c ) | ( ~b & d );
        int g = i;
        SHIFT;
    }
    for ( i = 16; i <= 31; ++i ) {
        uint32 f = ( d & b ) | ( ~d & c );
        int g = (5 * i + 1) & 0xf;  /* mod 16 */
        SHIFT;
    }
    for ( i = 32; i <= 47; ++i ) {
        uint32 f = b ^ c ^ d;
        int g = ( 3 * i + 5 ) & 0xf; /* mod 16 */
        SHIFT;
    }
    for ( i = 48; i <= 63; ++i ) {
        uint32 f= c ^ (b | ~d);
        int g = ( 7 * i ) & 0xf;
        SHIFT;
    }
    
    a = a + a0;
    
    /* Needed for debug only */
    /*
    b = b + b0;
    c = c + c0;
    d = d + d0;
    */
    if ( ( a & goalMask ) == 0 ) {
        atomicMin( goal, n ); 
        printf( "Goal! %d => %08x %08x %08x %08x\n", n, a, b, c, d );
    }
}

int hostShiftTable[64] = {  
    7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22,
    5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20,
    4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23,
    6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21 
};


uint32 hostKTable[64] = { 
    0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
    0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
    0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
    0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
    0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
    0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
    0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
    0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
    0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
    0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
    0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
    0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
    0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
    0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
    0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
    0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
};

#define checkCudaErrors(err) __checkCudaErrors( err, __FILE__, __LINE__ )

void __checkCudaErrors( hipError_t err, const char * file, const int line ) {
    if ( err != hipSuccess ) {
        fprintf( stderr, "Error %d (%s) at %s:%d\n",
                 err,
                 hipGetErrorString( err ),
                 file,
                 line );
        exit( 1 );
    }
}

int 
usage() {
    fprintf( stderr, "Usage: day4 <secret key> <num zeros>\n" );
    fprintf( stderr, "  Number of zeros must be 1-8.\n" );
    exit( 2 );
}

int 
main( int argc, char *argv[] ) {
    if ( argc < 2 || argc > 3 ) usage();

    char *fixed = argv[1];
    char fixedLen = strlen( fixed );
    uint32 mask;
    if ( argc == 3 ) {
        int numZeros = atoi( argv[2] );
        if ( numZeros < 1 || numZeros > 8 ) usage();
        switch ( numZeros ) {
        case 1: mask = 0x000000f0; break;
        case 2: mask = 0x000000ff; break;
        case 3: mask = 0x0000f0ff; break;
        case 4: mask = 0x0000ffff; break;
        case 5: mask = 0x00f0ffff; break;
        case 6: mask = 0x00ffffff; break;
        case 7: mask = 0xf0ffffff; break;
        case 8: mask = 0xffffffff; break;
        }
    }

    int maxSearch = 1000000000;
    int goal = maxSearch;

    /* Device-side pointers to tables and input */
    int *deviceShiftTable;
    uint32 *deviceKTable;
    char *deviceInput;
    int *output;

    size_t tableSize = 64 * sizeof( uint32 );
    
    checkCudaErrors( hipMalloc( &deviceShiftTable, tableSize ) );
    checkCudaErrors( hipMalloc( &deviceKTable, tableSize ) );
    checkCudaErrors( hipMalloc( &deviceInput, fixedLen ) );
    checkCudaErrors( hipMalloc( &output, sizeof( int ) ) );

    checkCudaErrors( hipMemcpy( deviceShiftTable, hostShiftTable, tableSize, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( deviceKTable, hostKTable, tableSize, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( deviceInput, fixed, fixedLen, hipMemcpyHostToDevice ) ); 
    checkCudaErrors( hipMemcpy( output, &maxSearch, sizeof( int ), hipMemcpyHostToDevice ) ); 
        
    /* My device has a maximum of 1024 threads per block and 
     * 2048 threads per multiprocessor.
     */
    int blockSize = 512;
    int numBlocks = 4;
    int stride = blockSize * numBlocks;
    int count = 100;
    for ( int start = 0; start < maxSearch && goal == maxSearch; start += stride ) {
        checkCudaErrors( hipGetLastError() );
        md5Kernel<<<numBlocks,blockSize>>>( deviceShiftTable, deviceKTable, deviceInput, fixedLen, mask, start, output );
        checkCudaErrors( hipGetLastError() );
        // slows us down by a factor of 2... not necessary when not using async?
        // checkCudaErrors( cudaDeviceSynchronize() );
        if ( count == 0 ) {
            checkCudaErrors( hipMemcpy( &goal, output, sizeof( int ), hipMemcpyDeviceToHost ) );
            count = 100;
        } else {
            --count;
        }
    }

    printf( "Answer: %d\n", goal );

    hipFree( deviceShiftTable );
    hipFree( deviceKTable );
    hipFree( deviceInput );
    
    hipDeviceReset();

    return 0;
}

 
